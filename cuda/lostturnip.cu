#include "hip/hip_runtime.h"
// -*-c++-*-
#include <chrono>
#include <iostream>
#include <sstream>

#include <lostturnip.hpp>

static void throw_cuda_error(const char *file, int line, hipError_t status) {
  std::stringstream msg;
  if (status == hipErrorUnknown) {
    msg << file << "(" << line << ") An Unknown CUDA Error Occurred :(";
  } else {
    msg << file << "(" << line << ") CUDA Error Occurred:\n" <<
      hipGetErrorString(status);
  }
#ifdef DUST_ENABLE_CUDA_PROFILER
  hipProfilerStop();
#endif
  throw std::runtime_error(msg.str());
}

static void handle_cuda_error(const char *file, int line,
                              hipError_t status = hipGetLastError()) {
#ifdef _DEBUG
  hipDeviceSynchronize();
#endif
  if (status != hipSuccess || (status = hipGetLastError()) != hipSuccess) {
    throw_cuda_error(file, line, status);
  }
}

#define CUDA_CALL( err ) (handle_cuda_error(__FILE__, __LINE__ , err))

__device__
float f(float c, float a, float b) {
  const auto g = [&](float x) { return (x + c) * (x - 1) * (x - 1) + 0.1f; };
  const float tol = 0.000001f;
  const auto ret = lostturnip::find_result(g, a, b, tol, 100);
  return ret.x;
}

__global__
void kernel(float *c, float *roots, size_t n_threads, size_t n_roots) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const float a = -4;
  const float b = 4.0f / 3.0f;
  if (i < n_roots) {
    roots[i] = f(c[i], a, b);
  }
}

void run(int n_roots, int n_threads) {
  const size_t blockSize = 128;
  const size_t blockCount = (n_threads + blockSize - 1) / blockSize;
  auto t0 = std::chrono::high_resolution_clock::now();

  // Generate a vector of parameters that we'll search over
  float *c_host = (float*)malloc(n_roots * sizeof(float));
  const float min = -2;
  const float max = 4;
  for (int i = 0; i < n_roots; ++i) {
    c_host[i] = (static_cast<float>(i) / (n_roots - 1)) * (max - min) + min;
  }

  float *roots;
  float *c;
  CUDA_CALL(hipMalloc((void**)&c, n_roots * sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&roots, n_roots * sizeof(float)));

  hipMemcpy(c, c_host, sizeof(float) * n_roots, hipMemcpyHostToDevice);

  kernel<<<blockCount, blockSize>>>(c, roots, n_threads, n_roots);

  CUDA_CALL(hipDeviceSynchronize());

  auto t1 = std::chrono::high_resolution_clock::now();

  std::chrono::duration<double> t = t1 - t0;

  float *roots_host = (float*) malloc(n_roots * sizeof(float));
  hipMemcpy(roots_host, roots, sizeof(float) * n_roots,
             hipMemcpyDeviceToHost);

  std::cout << "c,x" << std::endl;
  for (int i = 0; i < n_roots; ++i) {
    std::cout << c_host[i] << ", " << roots_host[i] << std::endl;
  }

  CUDA_CALL(hipFree(c));
  CUDA_CALL(hipFree(roots));
  free(c_host);
  free(roots_host);
}

int main(int argc, char *argv[]) {
  const int n_threads = 100;
  const int n_roots = 100;
  run(n_roots, n_threads);
  return 0;
}
